
#include <hip/hip_runtime.h>
﻿extern "C" {
    __global__ void Run(int n, float* __restrict input, float* __restrict output, float alpha) {
		int i = blockIdx.x*blockDim.x + threadIdx.x;
		if (i < n) output[i] =  input[i] > 0 ? input[i] : input[i] * alpha; 
	}
}