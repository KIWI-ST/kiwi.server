
#include <hip/hip_runtime.h>
﻿extern "C" {
    __global__ void Run(int n, float* __restrict left, float* __restrict right, float* __restrict output, int rightIsScalar) {
		int i = blockIdx.x*blockDim.x + threadIdx.x;
		if (i < n) {
            if (rightIsScalar == 1) { 
                output[i] = left[i] / right[0];
            } else {
                output[i] = left[i] / right[i];
            }
        }
	}
}