
#include <hip/hip_runtime.h>
﻿extern "C" {
    __global__ void Run(int n, float* __restrict x, float* __restrict output, int length, int offset, int inputCount) {
		int i = blockIdx.x*blockDim.x + threadIdx.x;
        
        int batch = i / length;
        int rest = i % length;

        int batchCount = n / length;

        int inputWidth = inputCount / batchCount;

		if (i < n) {
            output[i] = x[batch * inputWidth + rest + offset];
        }
	}
}