
#include <hip/hip_runtime.h>
﻿extern "C" {
    __global__ void Run(int n, double* __restrict input, double* __restrict output) {
		int i = blockIdx.x*blockDim.x + threadIdx.x;
		if (i < n) output[i] = log(input[i]);
	}
}